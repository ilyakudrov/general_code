#include "../../../lib/cpu/include/data.h"
// #include "../../../lib/gpu/include/general/reduction.h"
#include "../../../lib/gpu/include/observables/plaket.h"

// #include "hip/hip_runtime_api.h"

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

int x_size;
int y_size;
int z_size;
int t_size;

#define MATRIX_TYPE_CPU su3
#define MATRIX_TYPE_GPU matrix_gpu::su3

using namespace std;

#define NX 36
#define NY 36
#define NZ 36
#define NT 36
#define THREADS_PER_BLOCK 128
#define BLOCK_NUMBER NX *NY *NZ *NT / THREADS_PER_BLOCK

int main() {

  x_size = NX;
  y_size = NY;
  z_size = NZ;
  t_size = NT;

  int lattice_size = x_size * y_size * z_size * t_size;

  std::cout.precision(17);

  data<MATRIX_TYPE_CPU> conf;

  string conf_path = "../../confs/SU3_conf/gluodynamics/36^4/beta6.3/CONF0001";

  string conf_format = "double_qc2dstag";
  int bytes_skip = 0;
  bool convert = 0;

  get_data(conf, conf_path, conf_format, bytes_skip, convert);

  MATRIX_TYPE_GPU *dconf;
  gpuErrchk(hipMalloc(&dconf, lattice_size * 4 * sizeof(MATRIX_TYPE_GPU)));
  gpuErrchk(hipMemcpy(dconf, &conf.array[0],
                       lattice_size * 4 * sizeof(MATRIX_TYPE_GPU),
                       hipMemcpyHostToDevice));

  unsigned int *lat_indices =
      (unsigned int *)malloc(lattice_size * 9 * sizeof(unsigned int));
  int lattice_sizes[4] = {NX, NY, NZ, NT};
  plaket_gpu::make_plaket_indices(lat_indices, lattice_sizes);

  unsigned int *dlat_indices;
  gpuErrchk(hipMalloc(&dlat_indices, lattice_size * 9 * sizeof(unsigned int)));
  gpuErrchk(hipMemcpy(dlat_indices, lat_indices,
                       lattice_size * 9 * sizeof(unsigned int),
                       hipMemcpyHostToDevice));

  double *dtraces;
  gpuErrchk(hipMalloc(&dtraces, lattice_size * sizeof(double)));

  dim3 dimBlock(4, THREADS_PER_BLOCK / 4, 1);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // plaket
  hipEventRecord(start);

  // hipProfilerStart();

  plaket_gpu::plaket<<<BLOCK_NUMBER, THREADS_PER_BLOCK>>>(dconf, dtraces,
                                                          dlat_indices);

  // hipProfilerStop();

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket time " << milliseconds / 1000 << endl;

  double *traces = (double *)malloc(lattice_size * sizeof(double));
  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * sizeof(double),
                       hipMemcpyDeviceToHost));

  double plaket = 0;
  for (int i = 0; i < lattice_size; i++) {
    plaket += traces[i];
  }

  cout << "plaket aver " << plaket / lattice_size / 6 << endl;

  gpuErrchk(hipFree(dtraces));
  free(traces);

  // plaket_test1
  gpuErrchk(hipMalloc(&dtraces, lattice_size * 6 * sizeof(double)));
  hipEventRecord(start);

  plaket_gpu::plaket_test1<<<6 * BLOCK_NUMBER, THREADS_PER_BLOCK>>>(
      dconf, dtraces, dlat_indices);

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket_test1 time " << milliseconds / 1000 << endl;

  traces = (double *)malloc(lattice_size * 6 * sizeof(double));
  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * 6 * sizeof(double),
                       hipMemcpyDeviceToHost));

  plaket = 0;
  for (int i = 0; i < lattice_size * 6; i++) {
    plaket += traces[i];
  }

  cout << "plaket_test1 aver " << plaket / lattice_size / 6 << endl;

  gpuErrchk(hipFree(dtraces));
  free(traces);

  // plaket_test4

  gpuErrchk(hipMalloc(&dtraces, lattice_size * 6 * sizeof(double)));
  hipEventRecord(start);

  dim3 dimGrid(6, 16);
  cout << "number of blocks " << (NX * NY * NZ * NT * 6 + 6 * 16 - 1) / (6 * 16)
       << endl;

  plaket_gpu::plaket_test4<<<(NX * NY * NZ * NT * 6 + 6 * 16 - 1) / (6 * 16),
                             dimGrid>>>(dconf, dtraces, dlat_indices);

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket_test4 time " << milliseconds / 1000 << endl;

  traces = (double *)malloc(lattice_size * 6 * sizeof(double));
  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * 6 * sizeof(double),
                       hipMemcpyDeviceToHost));

  plaket = 0;
  for (int i = 0; i < lattice_size * 6; i++) {
    plaket += traces[i];
  }

  cout << "plaket_test4 aver " << plaket / lattice_size / 6 << endl;

  gpuErrchk(hipFree(dlat_indices));
  free(lat_indices);

  // plaket_test2

  lat_indices =
      (unsigned int *)malloc(lattice_size * 24 * sizeof(unsigned int));
  plaket_gpu::make_plaket_indices_test2(lat_indices, lattice_sizes);

  gpuErrchk(
      hipMalloc(&dlat_indices, lattice_size * 24 * sizeof(unsigned int)));
  gpuErrchk(hipMemcpy(dlat_indices, lat_indices,
                       lattice_size * 24 * sizeof(unsigned int),
                       hipMemcpyHostToDevice));

  hipEventRecord(start);

  plaket_gpu::plaket_test2<<<6 * BLOCK_NUMBER, THREADS_PER_BLOCK>>>(
      dconf, dtraces, dlat_indices);

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket_test2 time " << milliseconds / 1000 << endl;

  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * 6 * sizeof(double),
                       hipMemcpyDeviceToHost));

  plaket = 0;
  for (int i = 0; i < lattice_size * 6; i++) {
    plaket += traces[i];
  }

  cout << "plaket_test2 aver " << plaket / lattice_size / 6 << endl;

  free(lat_indices);
  gpuErrchk(hipFree(dlat_indices));
  gpuErrchk(hipFree(dconf));

  // plaket_test3

  unsigned int hsteps[4] = {4, 4 * NX, 4 * NX * NY, 4 * NX * NY * NZ};
  unsigned int harr_mu[6] = {0, 0, 0, 1, 1, 2};
  unsigned int harr_nu[6] = {1, 2, 3, 2, 3, 3};

  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(steps), hsteps, 4 * sizeof(unsigned int)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(arr_mu), harr_mu, 6 * sizeof(unsigned int)));
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(arr_nu), harr_nu, 6 * sizeof(unsigned int)));

  gpuErrchk(hipMalloc(&dconf, lattice_size * 4 * 2 * sizeof(MATRIX_TYPE_GPU)));
  conf.array.reserve(lattice_size * 4 * 2);
  for (int i = 0; i < lattice_size * 4; i++) {
    conf.array.push_back(conf.array[i]);
  }
  gpuErrchk(hipMemcpy(dconf, &conf.array[0],
                       lattice_size * 4 * 2 * sizeof(MATRIX_TYPE_GPU),
                       hipMemcpyHostToDevice));

  hipEventRecord(start);

  plaket_gpu::plaket_test3<<<6 * BLOCK_NUMBER, THREADS_PER_BLOCK>>>(dconf,
                                                                    dtraces);

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket_test3 time " << milliseconds / 1000 << endl;

  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * 6 * sizeof(double),
                       hipMemcpyDeviceToHost));

  plaket = 0;
  for (int i = 0; i < lattice_size * 6; i++) {
    plaket += traces[i];
  }

  cout << "plaket_test3 aver " << plaket / lattice_size / 6 << endl;

  gpuErrchk(hipFree(dtraces));
  free(traces);

  gpuErrchk(hipFree(dconf));

  // double *d_odata;
  // gpuErrchk(hipMalloc(&d_odata, BLOCK_NUMBER * sizeof(double)));

  // reduction::reduction<THREADS_PER_BLOCK>
  //     <<<BLOCK_NUMBER / 2, THREADS_PER_BLOCK>>>(dtraces, d_odata);

  // gpuErrchk(hipPeekAtLastError());
  // gpuErrchk(hipDeviceSynchronize());

  // double *odata = (double *)malloc(BLOCK_NUMBER * sizeof(double));
  // gpuErrchk(hipMemcpy((void *)odata, (const void *)d_odata, sizeof(double),
  //                      hipMemcpyDeviceToHost));
}