#include "../../../lib/cpu/include/data.h"
#include "../../../lib/gpu/include/general/reduction.h"
#include "../../../lib/gpu/include/observables/plaket.h"

#include "hip/hip_runtime_api.h"

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

int x_size;
int y_size;
int z_size;
int t_size;

#define MATRIX_TYPE_CPU su3
#define MATRIX_TYPE_GPU matrix_gpu::su3

using namespace std;

#define NX 36
#define NY 36
#define NZ 36
#define NT 36
#define THREADS_PER_BLOCK 128
#define BLOCK_NUMBER NX *NY *NZ *NT / THREADS_PER_BLOCK

int main() {

  x_size = 36;
  y_size = 36;
  z_size = 36;
  t_size = 36;

  int lattice_size = x_size * y_size * z_size * t_size;

  std::cout.precision(17);

  data<MATRIX_TYPE_CPU> conf;

  string conf_path = "../../confs/SU3_conf/gluodynamics/36^4/beta6.3/CONF0001";

  string conf_format = "double_qc2dstag";
  int bytes_skip = 0;
  bool convert = 0;

  get_data(conf, conf_path, conf_format, bytes_skip, convert);

  MATRIX_TYPE_GPU *dconf;
  gpuErrchk(hipMalloc(&dconf, lattice_size * 4 * sizeof(MATRIX_TYPE_GPU)));
  gpuErrchk(hipMemcpy(dconf, &conf.array[0],
                       lattice_size * 4 * sizeof(MATRIX_TYPE_GPU),
                       hipMemcpyHostToDevice));

  int *lat_indices = (int *)malloc(lattice_size * 9 * sizeof(int));
  int lattice_sizes[4] = {NX, NY, NZ, NT};
  plaket_gpu::make_plaket_indices(lat_indices, lattice_sizes);

  int *dlat_indices;
  gpuErrchk(hipMalloc(&dlat_indices, lattice_size * 9 * sizeof(int)));
  gpuErrchk(hipMemcpy(dlat_indices, lat_indices,
                       lattice_size * 9 * sizeof(int), hipMemcpyHostToDevice));

  double *dtraces;
  gpuErrchk(hipMalloc(&dtraces, lattice_size * sizeof(double)));

  dim3 dimBlock(4, THREADS_PER_BLOCK / 4, 1);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // plaket
  hipEventRecord(start);

  // hipProfilerStart();

  plaket_gpu::plaket<<<BLOCK_NUMBER, THREADS_PER_BLOCK>>>(dconf, dtraces,
                                                          dlat_indices);

  // hipProfilerStop();

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket time " << milliseconds / 1000 << endl;

  double *traces = (double *)malloc(lattice_size * sizeof(double));
  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * sizeof(double),
                       hipMemcpyDeviceToHost));

  double plaket = 0;
  for (int i = 0; i < lattice_size; i++) {
    plaket += traces[i];
  }

  cout << "plaket aver " << plaket / lattice_size / 6 << endl;

  gpuErrchk(hipFree(dtraces));
  free(traces);

  // plaket_test1
  gpuErrchk(hipMalloc(&dtraces, lattice_size * 6 * sizeof(double)));
  hipEventRecord(start);

  plaket_gpu::plaket_test1<<<6 * BLOCK_NUMBER, THREADS_PER_BLOCK>>>(
      dconf, dtraces, dlat_indices);

  hipEventRecord(stop);

  gpuErrchk(hipEventSynchronize(stop));
  milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  cout << "kernal plaket time " << milliseconds / 1000 << endl;

  traces = (double *)malloc(lattice_size * 6 * sizeof(double));
  gpuErrchk(hipMemcpy(traces, dtraces, lattice_size * 6 * sizeof(double),
                       hipMemcpyDeviceToHost));

  plaket = 0;
  for (int i = 0; i < lattice_size * 6; i++) {
    plaket += traces[i];
  }

  cout << "plaket aver " << plaket / lattice_size / 6 << endl;

  gpuErrchk(hipFree(dtraces));
  free(traces);

  // double *d_odata;
  // gpuErrchk(hipMalloc(&d_odata, BLOCK_NUMBER * sizeof(double)));

  // reduction::reduction<THREADS_PER_BLOCK>
  //     <<<BLOCK_NUMBER / 2, THREADS_PER_BLOCK>>>(dtraces, d_odata);

  // gpuErrchk(hipPeekAtLastError());
  // gpuErrchk(hipDeviceSynchronize());

  // double *odata = (double *)malloc(BLOCK_NUMBER * sizeof(double));
  // gpuErrchk(hipMemcpy((void *)odata, (const void *)d_odata, sizeof(double),
  //                      hipMemcpyDeviceToHost));
}